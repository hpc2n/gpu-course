//
// A simple CUDA program that adds two vectors together.
//
// Author: Mirko Myllykoski, Umeå University, 2019
//

#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hipblas.h>

int main(int argc, char const **argv)
{
    // read and validate the command line arguments

    if (argc < 2) {
        fprintf(stderr, "[error] No vector lenght was supplied.\n");
        return EXIT_FAILURE;
    }

    size_t n = atof(argv[1]);
    if (n < 1) {
        fprintf(stderr, "[error] The vector lenght was invalid.\n");
        return EXIT_FAILURE;
    }

    // allocate memory

    double *x;
    if (hipMallocManaged(&x, n*sizeof(double)) != hipSuccess) {
        fprintf(stderr, "[error] Failed to allocate memory for vector x.\n");
        return EXIT_FAILURE;
    }

    double *y;
    if (hipMallocManaged(&y, n*sizeof(double)) != hipSuccess) {
        fprintf(stderr, "[error] Failed to allocate memory for vector y.\n");
        return EXIT_FAILURE;
    }

    // initialize memory

    for (int i = 0; i < n; i++) {
        x[i] = 2.0 * rand()/RAND_MAX - 1.0;
        y[i] = 2.0 * rand()/RAND_MAX - 1.0;
    }

    // prefetch data to GPU memory

    int device = -1;
    if (hipGetDevice(&device) != hipSuccess) {
        fprintf(stderr, "[error] hipGetDevice() failed.\n");
        return EXIT_FAILURE;
    }

    if (hipMemPrefetchAsync(x, n*sizeof(double), device, NULL) != hipSuccess) {
        fprintf(stderr, "[error] hipMemPrefetchAsync() failed.\n");
        return EXIT_FAILURE;
    }

    if (hipMemPrefetchAsync(y, n*sizeof(double), device, NULL) != hipSuccess) {
        fprintf(stderr, "[error] hipMemPrefetchAsync() failed.\n");
        return EXIT_FAILURE;
    }

    if (hipDeviceSynchronize() != hipSuccess) {
        fprintf(stderr, "[error] hipDeviceSynchronize() failed.\n");
        return EXIT_FAILURE;
    }

    // initialize cuBLAS

    hipblasHandle_t handle;
    if (hipblasCreate(&handle) != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "[error] Failed to initialize cuBLAS.\n");
        return EXIT_FAILURE;
    }

    //
    // start timer
    //

    struct timespec start;
    clock_gettime(CLOCK_REALTIME, &start);

    //
    // compute y <- 2 * x + y
    //

    double alpha = 2.0;
    if (hipblasDaxpy(handle, n, &alpha, x, 1, y, 1) != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr,"[error] hipblasDaxpy() failed.\n");
        return EXIT_FAILURE;
    }

    //
    // wait until the hipblasDaxpy has finished
    //

    if (hipDeviceSynchronize() != hipSuccess) {
        fprintf(stderr, "[error] hipDeviceSynchronize() failed.\n");
        return EXIT_FAILURE;
    }

    //
    // stop timer and report
    //

    struct timespec stop;
    clock_gettime(CLOCK_REALTIME, &stop);

    double time =
        (stop.tv_sec - start.tv_sec) + (stop.tv_nsec - start.tv_nsec)*1E-9;

    printf("Runtime was %.3f s.\n", time);
    printf("Floprate was %.0f GFlops.\n", (2*n/time)*1E-9);
    printf("Memory throughput %.0f GB/s.\n", (3*n*sizeof(double)/time)*1E-9);

    // de-initialize cuBLAS

    if (hipblasDestroy(handle) != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "[error] hipblasDestroy() failed.\n");
        return EXIT_FAILURE;
    }

    // free the allocated memory

    if (hipFree(x) != hipSuccess) {
        fprintf(stderr, "[error] hipFree() failed.\n");
        return EXIT_FAILURE;
    }

    if (hipFree(y) != hipSuccess) {
        fprintf(stderr, "[error] hipFree() failed.\n");
        return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;
}
