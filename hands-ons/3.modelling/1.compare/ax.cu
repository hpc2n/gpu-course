#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#define CHECK_CUDA_ERROR(exp) {                     \
    hipError_t ret = (exp);                        \
    if (ret != hipSuccess) {                       \
        fprintf(stderr, "[error] %s:%d: %s (%s)\n", \
            __FILE__, __LINE__,                     \
            hipGetErrorName(ret),                  \
            hipGetErrorString(ret));               \
        exit(EXIT_FAILURE);                         \
    }                                               \
}

// a kernel that multiplies a vector y with a scalar alpha
__global__ void ax_kernel(int n, double alpha, double *y)
{
    //
    // Each thread is going to begin from the array element that matches it's
    // global index number. For blockDim.x = 4, gridDim.x 2, we have:
    // threadIdx.x : 0 1 2 3 0 1 2 3
    // blockIdx.x  : 0 0 0 0 1 1 1 1
    // blockDim.x  : 4 4 4 4 4 4 4 4
    // thread_id   : 0 1 2 3,4 5 6 7
    //
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_count = gridDim.x * blockDim.x;

    //
    // Each thread is going to jump over <grid dimension> * <block dimension>
    // array elements. For blockDim.x = 4, gridDim.x 2, we have:
    // 0 1 2 3,4 5 6 7|0 1 2 3,4 5 6 7|0 1 2 3,4 5 6 7|0 1 2 3,4 5 6 7|0 ...
    //
    for (int i = thread_id; i < n; i += thread_count)
        y[i] = alpha * y[i];
}

int main(int argc, char const **argv)
{
    double alpha = 2.0;

    // read and validate the command line arguments

    if (argc < 2) {
        fprintf(stderr, "[error] No vector length was supplied.\n");
        return EXIT_FAILURE;
    }

    int n = atof(argv[1]);
    if (n < 1) {
        fprintf(stderr, "[error] The vector length was invalid.\n");
        return EXIT_FAILURE;
    }
    
    srand(time(NULL));

    // allocate host memory for the vector and it's duplicate

    double *y, *_y;
    if ((y = (double *) malloc(n*sizeof(double))) == NULL) {
        fprintf(stderr,
            "[error] Failed to allocate host memory for vector y.\n");
        return EXIT_FAILURE;
    }
    if ((_y = (double *) malloc(n*sizeof(double))) == NULL) {
        fprintf(stderr,
            "[error] Failed to allocate host memory for vector _y.\n");
        return EXIT_FAILURE;
    }

    // initialize host memory and store a copy for a later validation

    for (int i = 0; i < n; i++)
        y[i] = _y[i] = 1.0*rand()/RAND_MAX;

    // allocate device memory

    double *d_y;
    CHECK_CUDA_ERROR(hipMalloc(&d_y, n*sizeof(double)));

    // copy the vector from the host memory to the device memory

    CHECK_CUDA_ERROR(
        hipMemcpy(d_y, y, n*sizeof(double), hipMemcpyHostToDevice));

    // start timer
    struct timespec ts_start;
    clock_gettime(CLOCK_MONOTONIC, &ts_start);

    // launch the kernel

    dim3 threads = 256;
    dim3 blocks = max(1, min(256, n/threads.x));
    ax_kernel<<<blocks, threads>>>(n, alpha, d_y);

    // wait until the device is ready and stop the timer
    
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    struct timespec ts_stop;
    clock_gettime(CLOCK_MONOTONIC, &ts_stop);

    // calculate metrics
    
    double time = ts_stop.tv_sec - ts_start.tv_sec +
        1.0e-9*(ts_stop.tv_nsec - ts_start.tv_nsec); 
    printf("Time = %f s\n", time);
    printf("Floprate = %.1f GFlops\n", 1.0E-9 * n / time);
    printf("Memory throughput = %.0f GB/s\n", 
        1.0E-9 * 2 * n * sizeof(double) / time);

    // free the allocated memory

    free(y); free(_y);
    CHECK_CUDA_ERROR(hipFree(d_y));
}
