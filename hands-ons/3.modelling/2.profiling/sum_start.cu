#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#define CHECK_CUDA_ERROR(exp) {                     \
    hipError_t ret = (exp);                        \
    if (ret != hipSuccess) {                       \
        fprintf(stderr, "[error] %s:%d: %s (%s)\n", \
            __FILE__, __LINE__,                     \
            hipGetErrorName(ret),                  \
            hipGetErrorString(ret));               \
        exit(EXIT_FAILURE);                         \
    }                                               \
}

// in order to keep things simple, the thread block size is fixed
#define THREAD_BLOCK_SIZE 128

// a function that returns the ceil of a/b. That is,
//     DIVCEIL(5, 2) = ceil(5/2) = ceil(2.5) = 3.
static int DIVCEIL(int a, int b)
{
    return (a+b-1)/b;
}

//
// A kernel that partially sums together the elements of a vector x. The partial
// sums are stored to a vector y such that
//     y[i] = x[i] + x[i+thread_count] + x[i+2*thread_count] + ...
//
__global__ void partial_sum_kernel(int n, double const *x, double *y)
{
    int thread_id = blockIdx.x * THREAD_BLOCK_SIZE + threadIdx.x;
    int thread_count = gridDim.x * THREAD_BLOCK_SIZE;

    double v = 0.0;
    for (int i = thread_id; i < n; i += thread_count)
        v += x[i];
    
    y[thread_id] = v;
}

// a function that sums together the elements of a vector x
double final_sum(int n, double *x)
{
    double v = 0;
    for (int i = 0; i < n; i++)
        v += x[i];
    return v;
}

int main(int argc, char **argv)
{
    // read and validate the command line arguments

    if (argc < 2) {
        fprintf(stderr, "[error] No vector length was supplied.\n");
        return EXIT_FAILURE;
    }
    
    if (argc < 3) {
        fprintf(stderr, 
            "[error] No intermediate vector length was supplied.\n");
        return EXIT_FAILURE;
    }

    int n = atof(argv[1]);
    if (n < 1) {
        fprintf(stderr, "[error] The vector length was invalid.\n");
        return EXIT_FAILURE;
    }
    int m = atof(argv[2]);
    if (m < 1) {
        fprintf(stderr, 
            "[error] The intermediate vector length was invalid.\n");
        return EXIT_FAILURE;
    }
    
    // in order to keep things simple, m is converted to a multiple of
    // THREAD_BLOCK_SIZE
    m = DIVCEIL(m, THREAD_BLOCK_SIZE)*THREAD_BLOCK_SIZE;
        
    srand(time(NULL));
    
    // allocate host memory for the vectors y and x
    
    double *y, *x;
    if ((y = (double *) malloc(m*sizeof(double))) == NULL) {
        fprintf(stderr,
            "[error] Failed to allocate host memory for vector y.\n");
        return EXIT_FAILURE;
    }
    if ((x = (double *) malloc(n*sizeof(double))) == NULL) {
        fprintf(stderr,
            "[error] Failed to allocate host memory for vector x.\n");
        return EXIT_FAILURE;
    }
    
    // initialize host memory

    for (int i = 0; i < n; i++)
        x[i] = 2.0*rand()/RAND_MAX - 1.0;

    // allocate device memory

    double *d_y, *d_x;
    CHECK_CUDA_ERROR(hipMalloc(&d_y, m*sizeof(double)));
    CHECK_CUDA_ERROR(hipMalloc(&d_x, n*sizeof(double)));
    
    // copy the vector x from the host memory to the device memory

    CHECK_CUDA_ERROR(
        hipMemcpy(d_x, x, n*sizeof(double), hipMemcpyHostToDevice));

    // launch the kernel, note that m is a multiple of THREAD_BLOCK_SIZE
    
    partial_sum_kernel<<<m/THREAD_BLOCK_SIZE, THREAD_BLOCK_SIZE>>>(n, d_x, d_y);
    
    // copy the vector y from the device memory to the host memory
    
    CHECK_CUDA_ERROR(
        hipMemcpy(y, d_y, m*sizeof(double), hipMemcpyDeviceToHost));
    
    // compute the final sum
    
    double sum = final_sum(m, y);

    // validate the result (Kahan)
    
    double sum2 = 0.0, c = 0.0;
    for (int i = 0; i < n; i++) {
        double z = x[i] - c;
        double t = sum2 + z;
        c = (t - sum2) - z;
        sum2 = t;
    }
    printf("Residual = %e\n", fabs(sum2 - sum)/fabs(sum2));
    
    // free the allocated memory
    
    free(y); free(x);
    CHECK_CUDA_ERROR(hipFree(d_y));
    CHECK_CUDA_ERROR(hipFree(d_x));

    return EXIT_SUCCESS;
}
