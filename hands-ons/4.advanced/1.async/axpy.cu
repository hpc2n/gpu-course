#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#define CHECK_CUDA_ERROR(exp) {                     \
    hipError_t ret = (exp);                        \
    if (ret != hipSuccess) {                       \
        fprintf(stderr, "[error] %s:%d: %s (%s)\n", \
            __FILE__, __LINE__,                     \
            hipGetErrorName(ret),                  \
            hipGetErrorString(ret));               \
        exit(EXIT_FAILURE);                         \
    }                                               \
}

// a kernel that computes the AXPY operation
__global__ void axpy_kernel(int n, double alpha, double *x, double *y)
{
    //
    // Each thread is going to begin from the array element that matches its
    // global index number. For blockDim.x = 4, gridDim.x 2, we have:
    // threadIdx.x : 0 1 2 3 0 1 2 3
    // blockIdx.x  : 0 0 0 0 1 1 1 1
    // blockDim.x  : 4 4 4 4 4 4 4 4
    // thread_id   : 0 1 2 3,4 5 6 7
    //
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_count = gridDim.x * blockDim.x;

    //
    // Each thread is going to jump over <grid dimension> * <block dimension>
    // array elements. For blockDim.x = 4, gridDim.x 2, we have:
    // 0 1 2 3,4 5 6 7|0 1 2 3,4 5 6 7|0 1 2 3,4 5 6 7|0 1 2 3,4 5 6 7|0 ...
    //
    for (int i = thread_id; i < n; i += thread_count)
        y[i] = alpha * x[i] + y[i];
}

int main(int argc, char const **argv)
{
    double alpha = 2.0;

    // read and validate the command line arguments

    if (argc < 2) {
        fprintf(stderr, "[error] No vector length was supplied.\n");
        return EXIT_FAILURE;
    }

    int n = atof(argv[1]);
    if (n < 1) {
        fprintf(stderr, "[error] The vector length was invalid.\n");
        return EXIT_FAILURE;
    }
    
    srand(time(NULL));

    // allocate host memory for the vectors and the duplicate

    double *x, *y, *_y;
    if ((x = (double *) malloc(n*sizeof(double))) == NULL) {
        fprintf(stderr,
            "[error] Failed to allocate host memory for vector x.\n");
        return EXIT_FAILURE;
    }
    if ((y = (double *) malloc(n*sizeof(double))) == NULL) {
        fprintf(stderr,
            "[error] Failed to allocate host memory for vector y.\n");
        return EXIT_FAILURE;
    }
    if ((_y = (double *) malloc(n*sizeof(double))) == NULL) {
        fprintf(stderr,
            "[error] Failed to allocate host memory for vector _y.\n");
        return EXIT_FAILURE;
    }

    // initialize host memory and store a copy for a later validation

    for (int i = 0; i < n; i++) {
        x[i] = 1.0*rand()/RAND_MAX;
        y[i] = _y[i] = 1.0*rand()/RAND_MAX;
    }

    // allocate device memory

    double *d_y, *d_x;
    CHECK_CUDA_ERROR(hipMalloc(&d_x, n*sizeof(double)));
    CHECK_CUDA_ERROR(hipMalloc(&d_y, n*sizeof(double)));

    // start timer
    
    struct timespec start, stop;
    clock_gettime(CLOCK_REALTIME, &start);
    
    // copy the vector from the host memory to the device memory

    CHECK_CUDA_ERROR(
        hipMemcpy(d_x, x, n*sizeof(double), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(
        hipMemcpy(d_y, y, n*sizeof(double), hipMemcpyHostToDevice));
    
    // launch the kernel

    dim3 threads = 256;
    dim3 blocks = max(1, min(256, n/threads.x));
    axpy_kernel<<<blocks, threads>>>(n, alpha, d_x, d_y);

    CHECK_CUDA_ERROR(hipGetLastError());

    // copy the vector from the device memory to the host memory

    CHECK_CUDA_ERROR(
        hipMemcpy(y, d_y, n*sizeof(double), hipMemcpyDeviceToHost));
    
    // stop timer
    
    clock_gettime(CLOCK_REALTIME, &stop);

    double time =
        (stop.tv_sec - start.tv_sec) + (stop.tv_nsec - start.tv_nsec)*1E-9;

    printf("Runtime was %f seconds.\n", time);

    // validate the result by computing sqrt((x-alpha*_x)^2)

    double res = 0.0;
    
    for (int i = 0; i < n; i++)
        res +=
            (y[i] - (alpha * x[i] + _y[i])) * (y[i] - (alpha * x[i] + _y[i]));
    
    printf("Residual = %e\n", sqrt(res));

    // free the allocated memory

    free(x), free(y); free(_y);
    CHECK_CUDA_ERROR(hipFree(d_x));
    CHECK_CUDA_ERROR(hipFree(d_y));
}
