#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#define CHECK_CUDA_ERROR(exp) {                     \
    hipError_t ret = (exp);                        \
    if (ret != hipSuccess) {                       \
        fprintf(stderr, "[error] %s:%d: %s (%s)\n", \
            __FILE__, __LINE__,                     \
            hipGetErrorName(ret),                  \
            hipGetErrorString(ret));               \
        exit(EXIT_FAILURE);                         \
    }                                               \
}

// a kernel that compute the AXPY operation
__global__ void axpy_kernel(int n, double alpha, double *x, double *y)
{
    //
    // Each thread is going to begin from the array element that matches it's
    // global index number. For blockDim.x = 4, gridDim.x 2, we have:
    // threadIdx.x : 0 1 2 3 0 1 2 3
    // blockIdx.x  : 0 0 0 0 1 1 1 1
    // blockDim.x  : 4 4 4 4 4 4 4 4
    // thread_id   : 0 1 2 3,4 5 6 7
    //
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_count = gridDim.x * blockDim.x;

    //
    // Each thread is going to jump over <grid dimension> * <block dimension>
    // array elements. For blockDim.x = 4, gridDim.x 2, we have:
    // 0 1 2 3,4 5 6 7|0 1 2 3,4 5 6 7|0 1 2 3,4 5 6 7|0 1 2 3,4 5 6 7|0 ...
    //
    for (int i = thread_id; i < n; i += thread_count)
        y[i] = alpha * x[i] + y[i];
}

int main(int argc, char const **argv)
{
    double alpha = 2.0;

    // read and validate the command line arguments

    if (argc < 2) {
        fprintf(stderr, "[error] No vector length was supplied.\n");
        return EXIT_FAILURE;
    }

    int n = atof(argv[1]);
    if (n < 1) {
        fprintf(stderr, "[error] The vector length was invalid.\n");
        return EXIT_FAILURE;
    }
    
    srand(time(NULL));
    
    // create a stream
    
    hipStream_t stream;
    CHECK_CUDA_ERROR(hipStreamCreate(&stream));

    // allocate host memory for the vectors and the duplicate

    double *x, *y, *_y;
    if ((x = (double *) malloc(n*sizeof(double))) == NULL) {
        fprintf(stderr,
            "[error] Failed to allocate host memory for vector x.\n");
        return EXIT_FAILURE;
    }
    if ((y = (double *) malloc(n*sizeof(double))) == NULL) {
        fprintf(stderr,
            "[error] Failed to allocate host memory for vector y.\n");
        return EXIT_FAILURE;
    }
    if ((_y = (double *) malloc(n*sizeof(double))) == NULL) {
        fprintf(stderr,
            "[error] Failed to allocate host memory for vector _y.\n");
        return EXIT_FAILURE;
    }
    
    // pin the vectors x and y to the host memory
    
    CHECK_CUDA_ERROR(
        hipHostRegister(x, n*sizeof(double), hipHostRegisterDefault));
    CHECK_CUDA_ERROR(
        hipHostRegister(y, n*sizeof(double), hipHostRegisterDefault));

    // initialize host memory and store a copy for a later validation

    for (int i = 0; i < n; i++) {
        x[i] = 1.0*rand()/RAND_MAX;
        y[i] = _y[i] = 1.0*rand()/RAND_MAX;
    }

    // allocate device memory

    double *d_y, *d_x;
    CHECK_CUDA_ERROR(hipMalloc(&d_x, n*sizeof(double)));
    CHECK_CUDA_ERROR(hipMalloc(&d_y, n*sizeof(double)));
    
    // start timer
    
    hipEvent_t start;
    CHECK_CUDA_ERROR(hipEventCreate(&start));
    CHECK_CUDA_ERROR(hipEventRecord (start, stream));

    // copy the vector from the host memory to the device memory

    CHECK_CUDA_ERROR(
        hipMemcpyAsync(
            d_x, x, n*sizeof(double), hipMemcpyHostToDevice, stream));
    CHECK_CUDA_ERROR(
        hipMemcpyAsync(
            d_y, y, n*sizeof(double), hipMemcpyHostToDevice, stream));
    
    // launch the kernel

    dim3 threads = 256;
    dim3 blocks = max(1, min(256, n/threads.x));
    axpy_kernel<<<blocks, threads, 0, stream>>>(n, alpha, d_x, d_y);

    CHECK_CUDA_ERROR(hipGetLastError());

    // copy the vector from the device memory to the host memory

    CHECK_CUDA_ERROR(
        hipMemcpyAsync(
            y, d_y, n*sizeof(double), hipMemcpyDeviceToHost, stream));

    // stop timer

    hipEvent_t stop;
    CHECK_CUDA_ERROR(hipEventCreate(&stop));
    CHECK_CUDA_ERROR(hipEventRecord (stop, stream));
    
    // wait until the stream is empty
    CHECK_CUDA_ERROR(hipStreamSynchronize(stream));
    
    // report run time
    
    float time;
    CHECK_CUDA_ERROR(hipEventElapsedTime(&time, start, stop));
    printf("Runtime was %f seconds.\n", 1E-3*time);

    // validate the result by computing sqrt((x-alpha*_x)^2)

    double res = 0.0;
    
    for (int i = 0; i < n; i++)
        res +=
            (y[i] - (alpha * x[i] + _y[i])) * (y[i] - (alpha * x[i] + _y[i]));
    
    printf("Residual = %e\n", sqrt(res));

    // free the allocated memory

    free(x), free(y); free(_y);
    CHECK_CUDA_ERROR(hipEventDestroy (start));
    CHECK_CUDA_ERROR(hipEventDestroy (stop));
    CHECK_CUDA_ERROR(hipFree(d_x));
    CHECK_CUDA_ERROR(hipFree(d_y));
}
