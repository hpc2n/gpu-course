
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define CHECK_CUDA_ERROR(exp) {                     \
    hipError_t ret = (exp);                        \
    if (ret != hipSuccess) {                       \
        fprintf(stderr, "[error] %s:%d: %s (%s)\n", \
            __FILE__, __LINE__,                     \
            hipGetErrorName(ret),                  \
            hipGetErrorString(ret));               \
        exit(EXIT_FAILURE);                         \
    }                                               \
}

// a kernel that prints the contents of an array
__global__ void print_array(int n, int *array)
{
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < n)
        printf("array[%d] = %d\n", thread_id, array[thread_id]);
}

int main(int argc, char const **argv)
{
    hipError_t ret;
    
    // array size
    int n = 32;
    
    // initialize host memory
    int *host = (int*) malloc(n*sizeof(int));
    for (int i = 0; i < n; i++)
        host[i] = i;

    // allocate device memory
    int *device;
    ret = hipMalloc(&device, n*sizeof(int));
    if (ret != hipSuccess) {
        printf("Function hipMalloc failed\n");
        printf("Error code: %s\n", hipGetErrorName(ret));
        printf("Error message: %s\n", hipGetErrorString(ret));
        exit(EXIT_FAILURE);
    }
    
    // move data from **host to device**
    //ret = cudaMemcpy(device, host, n*sizeof(int), cudaMemcpyDeviceToHost);
    ret = hipMemcpy(device, host, n*sizeof(int), hipMemcpyHostToDevice);
    if (ret != hipSuccess) {
        printf("Function hipMemcpy failed\n");
        printf("Error code: %s\n", hipGetErrorName(ret));
        printf("Error message: %s\n", hipGetErrorString(ret));
        exit(EXIT_FAILURE);
    }

    // call the kernel
    dim3 threads = 32;
    dim3 blocks = (n+threads.x-1)/threads.x; 
    print_array<<<blocks, threads>>>(n, device);
    
    ret = hipGetLastError();
    if (ret != hipSuccess) {
        printf("Kernel launch failed\n");
        printf("Error code: %s\n", hipGetErrorName(ret));
        printf("Error message: %s\n", hipGetErrorString(ret));
        exit(EXIT_FAILURE);
    }

    // wait until the GPU has executed the kernel
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    
    return EXIT_SUCCESS;
}
