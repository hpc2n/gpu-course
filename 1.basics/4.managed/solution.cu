
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hipblas.h>

#define CHECK_CUDA_ERROR(exp) {                     \
    hipError_t ret = (exp);                        \
    if (ret != hipSuccess) {                       \
        fprintf(stderr, "[error] %s:%d: %s (%s)\n", \
            __FILE__, __LINE__,                     \
            hipGetErrorName(ret),                  \
            hipGetErrorString(ret));               \
        exit(EXIT_FAILURE);                         \
    }                                               \
}

#define CHECK_CUBLAS_ERROR(exp) {                   \
    hipblasStatus_t ret = (exp);                     \
    if (ret != HIPBLAS_STATUS_SUCCESS) {             \
        fprintf(stderr,                             \
            "[error] %s:%d: cuBLAS error\n",        \
            __FILE__, __LINE__);                    \
        exit(EXIT_FAILURE);                         \
    }                                               \
}


int main(int argc, char const **argv)
{
    // read and validate the command line arguments

    if (argc < 2) {
        fprintf(stderr, "[error] No vector lenght was supplied.\n");
        return EXIT_FAILURE;
    }

    int n = atof(argv[1]);
    if (n < 1) {
        fprintf(stderr, "[error] The vector lenght was invalid.\n");
        return EXIT_FAILURE;
    }
    
    srand(time(NULL));

    // allocate memory

    double *x, *y, *_y;
    CHECK_CUDA_ERROR(hipMallocManaged(&x, n*sizeof(double)));
    CHECK_CUDA_ERROR(hipMallocManaged(&y, n*sizeof(double)));
    if ((_y = (double *) malloc(n*sizeof(double))) == NULL) {
        fprintf(stderr,
            "[error] Failed to allocate host memory for vector _y.\n");
        return EXIT_FAILURE;
    }

    // initialize memory

    for (int i = 0; i < n; i++) {
        x[i] = 2.0 * rand()/RAND_MAX - 1.0;
        y[i] = _y[i] = 2.0 * rand()/RAND_MAX - 1.0;
    }

    hipblasHandle_t handle;
    CHECK_CUBLAS_ERROR(hipblasCreate(&handle));
    
    // compute y <- 2 * x + y

    double alpha = 2.0;
    CHECK_CUBLAS_ERROR(hipblasDaxpy(handle, n, &alpha, x, 1, y, 1));
    
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    // validate the result

    double res = 0.0;
    for (int i = 0; i < n; i++)
        res +=
            (y[i] - (alpha * x[i] + _y[i])) * (y[i] - (alpha * x[i] + _y[i]));
    printf("Residual = %e\n", sqrt(res));

    // free the allocated memory

    CHECK_CUBLAS_ERROR(hipblasDestroy(handle));
    
    CHECK_CUDA_ERROR(hipFree(x));
    CHECK_CUDA_ERROR(hipFree(y));
    free(_y);

    return EXIT_SUCCESS;
}
