#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#define CHECK_CUDA_ERROR(exp) {                     \
    hipError_t ret = (exp);                        \
    if (ret != hipSuccess) {                       \
        fprintf(stderr, "[error] %s:%d: %s (%s)\n", \
            __FILE__, __LINE__,                     \
            hipGetErrorName(ret),                  \
            hipGetErrorString(ret));               \
        exit(EXIT_FAILURE);                         \
    }                                               \
}

// a kernel that multiplies a vector with a scalar
__global__ void ax_kernel(int n, double alpha, double *x)
{
    //
    // Each thread is going to begin from the array element matches it's own
    // global index number. For blockDim.x = 4, gridDim.x 2, we have:
    // threadIdx.x : 0 1 2 3 0 1 2 3
    // blockIdx.x  : 0 0 0 0 1 1 1 1
    // blockDim.x  : 4 4 4 4 4 4 4 4
    // thread_id   : 0 1 2 3,4 5 6 7
    //
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_count = gridDim.x * blockDim.x;

    //
    // Each thread is going to jump over <grid dimension> * <block dimension>
    // array elements. For blockDim.x = 4, gridDim.x 2, we have:
    // 0 1 2 3,4 5 6 7|0 1 2 3,4 5 6 7|0 1 2 3,4 5 6 7|0 1 2 3,4 5 6 7|0 ...
    //
    for (int i = thread_id; i < n; i += thread_count)
        x[thread_id] = alpha * x[thread_id];
}

int main(int argc, char const **argv)
{
    double alpha = 2.0;

    // read and validate the command line arguments

    if (argc < 2) {
        fprintf(stderr, "[error] No vector length was supplied.\n");
        return EXIT_FAILURE;
    }

    int n = atof(argv[1]);
    if (n < 1) {
        fprintf(stderr, "[error] The vector length was invalid.\n");
        return EXIT_FAILURE;
    }
    
    srand(time(NULL));

    // allocate host memory for the vector and it's duplicate

    double *x, *_x;
    if ((x = (double *) malloc(n*sizeof(double))) == NULL) {
        fprintf(stderr,
            "[error] Failed to allocate host memory for vector x.\n");
        return EXIT_FAILURE;
    }
    if ((_x = (double *) malloc(n*sizeof(double))) == NULL) {
        fprintf(stderr,
            "[error] Failed to allocate host memory for vector _x.\n");
        return EXIT_FAILURE;
    }

    // initialize host memory and store a copy for a later validation

    for (int i = 0; i < n; i++)
        x[i] = _x[i] = 1.0*rand()/RAND_MAX;

    // allocate device memory

    double *d_x;
    CHECK_CUDA_ERROR(hipMalloc(&d_x, n*sizeof(double)));

    // copy the vector from the host memory to the device memory

    CHECK_CUDA_ERROR(
        hipMemcpy(d_x, x, n*sizeof(double), hipMemcpyHostToDevice));

    // launch the kernel

    dim3 threads = 128;
    dim3 blocks = min(n, 64*threads.x);
    ax_kernel<<<blocks, threads>>>(n, alpha, d_x);

    CHECK_CUDA_ERROR(hipGetLastError());

    // copy the vector from the device memory to the host memory

    CHECK_CUDA_ERROR(
        hipMemcpy(x, d_x, n*sizeof(double), hipMemcpyDeviceToHost));

    // validate the result by computing sqrt((x-alpha*_x)^2)

    double res = 0.0;
    
    for (int i = 0; i < n; i++)
        res += (x[i]-alpha*_x[i]) * (x[i]-alpha*_x[i]);
    
    // A little assistance: comment out the above for loop and use this one
    // to validate the output of the AXPY operation.
    // for (int i = 0; i < n; i++)
    //    res +=
    //        (y[i] - (alpha * x[i] + _y[i])) * (y[i] - (alpha * x[i] + _y[i]));
    
    printf("Residual = %e\n", sqrt(res));

    // free the allocated memory

    free(x); free(_x);
    CHECK_CUDA_ERROR(hipFree(d_x));
}
